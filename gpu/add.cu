#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vector_add_kernel(float *A, float *B, float *C, int n)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < n)
  {
    C[i] = A[i] + B[i] + 0.0f;
  }
}

extern "C"  void  vector_add_gpu(float *A, float *B, float *C, int n)
{
  dim3 grid, block;

  block.x = 256;
  grid.x = (n + block.x - 1) / block.x;
  printf("CUDA kernel launch with %d blocks of %d threads\n", grid.x, block.x);

  vector_add_kernel<<<grid, block>>>(A, B, C, n);
}
