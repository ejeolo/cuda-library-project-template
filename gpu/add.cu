#include <stdio.h>
#include <hip/hip_runtime.h>

#include "inc/add.cuh"

// different name in this level for different typename, as extern "C" can not decorate template function that is in C++;

extern "C"  void  vector_add_gpu_s(float *A, float *B, float *C, int n)
{
  dim3 grid, block;

  block.x = 256;
  grid.x = (n + block.x - 1) / block.x;
  printf("CUDA kernel launch with %d blocks of %d threads\n", grid.x, block.x);

  vector_add_kernel<><<<grid, block>>>(A, B, C, n);
}

extern "C"  void  vector_add_gpu_d(double* A, double* B, double* C, int n)
{
  dim3 grid, block;

  block.x = 256;
  grid.x = (n + block.x - 1) / block.x;
  printf("CUDA kernel launch with %d blocks of %d threads\n", grid.x, block.x);

  vector_add_kernel<><<<grid, block>>>(A, B, C, n);
}
