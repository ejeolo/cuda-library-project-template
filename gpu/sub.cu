#include <stdio.h>
#include <hip/hip_runtime.h>
#include "inc/sub.cuh"

extern "C"  void  vector_sub_gpu_s(float *A, float *B, float *C, int n)
{
  dim3 grid, block;

  block.x = 256;
  grid.x = (n + block.x - 1) / block.x;
  printf("CUDA kernel launch with %d blocks of %d threads\n", grid.x, block.x);

  vector_sub_kernel<><<<grid, block>>>(A, B, C, n);
}

extern "C"  void  vector_sub_gpu_d(double *A, double *B, double *C, int n)
{
  dim3 grid, block;

  block.x = 256;
  grid.x = (n + block.x - 1) / block.x;
  printf("CUDA kernel launch with %d blocks of %d threads\n", grid.x, block.x);

  vector_sub_kernel<><<<grid, block>>>(A, B, C, n);
}
